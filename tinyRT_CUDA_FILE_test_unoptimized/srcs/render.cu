#include "hip/hip_runtime.h"

#include "tinyRT.hpp"
#include <fstream>

void initSDL() {
	std::cout << "Starting render" << std::endl;
}

void quitSDL() {
    std::cout << "end render" << std::endl;
}

void render(Vector3 *pix, int s) {
	fstream myfile;
	std::cout << "iter " << s << std::endl;
	myfile.open("render.ppm", fstream::out);
	myfile << "P3\n" << W << " " << H << "\n255\n";
	for (int col = 0; col < W; col++) {
		for (int row = 0; row < H; row++) {
			int pixel_index = row*W + col;
			myfile << std::min((int)(pix[pixel_index].x / (float)(s + 1)), 255) << " ";
			myfile << std::min((int)(pix[pixel_index].y / (float)(s + 1)), 255) << " ";
			myfile << std::min((int)(pix[pixel_index].z / (float)(s + 1)), 255) << std::endl;
		}
	}
}