#include "hip/hip_runtime.h"

#define inf 1e9
#define eps 1e-6

#include <fstream>
#include <iostream>


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <limits.h>
#include <ctime>
#include <vector>
#include <string>
#include <unordered_map>
#include <random>
#include <cstdint>
#include <algorithm>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <time.h>
#include "hip/hip_runtime.h"
#include ""
#include <thrust/sort.h>

#include <stdio.h>
#include <cmath>

#define DEVICE __host__ __device__

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}


struct Vec {
public:
	double x, y, z;
	DEVICE Vec(double x0, double y0, double z0) : x(x0), y(y0), z(z0) {}
	DEVICE Vec() : x(0), y(0), z(0) {}
	DEVICE Vec(double d) : x(d), y(d), z(d) {}
	DEVICE Vec operator+(const Vec& b) const { return Vec(x + b.x, y + b.y, z + b.z); }
	DEVICE Vec operator-(const Vec& b) const { return Vec(x - b.x, y - b.y, z - b.z); }
	DEVICE Vec operator*(double b) const { return Vec(x * b, y * b, z * b); }
	DEVICE Vec operator/(double b) const { return Vec(x / b, y / b, z / b); }
	DEVICE Vec operator*(const Vec& b) const { return Vec(x * b.x, y * b.y, z * b.z); }
	DEVICE Vec& norm() { return *this = *this / length(); }
	DEVICE double length() { return sqrt(x * x + y * y + z * z); }
	DEVICE double dot(const Vec& b) const { return x * b.x + y * b.y + z * b.z; }
	DEVICE Vec operator%(Vec& b) const { return Vec(y * b.z - z * b.y, z * b.x - x * b.z, x * b.y - y * b.x); }
	DEVICE bool operator!=(const Vec& b) const { return (b.x != x || b.y != y || b.z != z); }
};

struct Ray {
	Vec o, d;
	DEVICE Ray(Vec o0 = 0, Vec d0 = 0) { o = o0, d = d0.norm(); }
};

#include <algorithm>

struct AABB {
	DEVICE inline AABB() { min = Vec(inf, inf, inf); max = Vec(-inf, -inf, -inf); }	// an empty interval
	DEVICE inline AABB(Vec min_, Vec max_) { min = min_; max = max_; }
	DEVICE inline bool unbounded() const { return min.x == -inf || min.y == -inf || min.z == -inf || max.x == inf || max.y == inf || max.z == inf; }
	DEVICE inline size_t largestDimension() const {
		double dx = abs(max.x - min.x);
		double dy = abs(max.y - min.y);
		double dz = abs(max.z - min.z);
		if (dx > dy && dx > dz) {
			return 0;
		}
		if (dy > dz) {
			return 1;
		}
		return 2;
	}

	// ray-slab tests, see PBRT 2nd edition, section 4.2.1
	DEVICE inline bool intersect(const Ray& ray, const Vec& inverseDirection, double closestKnownT) const {
		bool xDirNegative = ray.d.x < 0;
		bool yDirNegative = ray.d.y < 0;
		bool zDirNegative = ray.d.z < 0;

		// check for ray intersection against x and y slabs
		float tmin = ((xDirNegative ? max.x : min.x) - ray.o.x) * inverseDirection.x;
		float tmax = ((xDirNegative ? min.x : max.x) - ray.o.x) * inverseDirection.x;
		float tymin = ((yDirNegative ? max.y : min.y) - ray.o.y) * inverseDirection.y;
		float tymax = ((yDirNegative ? min.y : max.y) - ray.o.y) * inverseDirection.y;
		if (tmin > tymax || tymin > tmax) {
			return false;
		}
		if (tymin > tmin) {
			tmin = tymin;
		}
		if (tymax < tmax) {
			tmax = tymax;
		}

		// check for ray intersection against z slab
		float tzmin = ((zDirNegative ? max.z : min.z) - ray.o.z) * inverseDirection.z;
		float tzmax = ((zDirNegative ? min.z : max.z) - ray.o.z) * inverseDirection.z;
		if (tmin > tzmax || tzmin > tmax) {
			return false;
		}
		if (tzmin > tmin) {
			tmin = tzmin;
		}
		if (tzmax < tmax) {
			tmax = tzmax;
		}
		return (tmin < closestKnownT) && (tmax > eps);
	}

	Vec min;
	Vec max;
};

DEVICE double mini(double a, double b) {
	if (a < b)
		return a;
	return b;
}


DEVICE double maxi(double a, double b) {
	if (a > b)
		return a;
	return b;
}

DEVICE inline AABB enclose(const AABB& firstBoundingBox, const AABB& secondBoundingBox) {
	AABB ret;

	ret.min.x = mini(firstBoundingBox.min.x, secondBoundingBox.min.x);
	ret.min.y = mini(firstBoundingBox.min.y, secondBoundingBox.min.y);
	ret.min.z = mini(firstBoundingBox.min.z, secondBoundingBox.min.z);

	ret.max.x = maxi(firstBoundingBox.max.x, secondBoundingBox.max.x);
	ret.max.y = maxi(firstBoundingBox.max.y, secondBoundingBox.max.y);
	ret.max.z = maxi(firstBoundingBox.max.z, secondBoundingBox.max.z);

	return ret;
}

DEVICE inline AABB enclose(const AABB& boundingBox, const Vec& point) {
	AABB ret;

	ret.min.x = mini(boundingBox.min.x, point.x);
	ret.min.y = mini(boundingBox.min.y, point.y);
	ret.min.z = mini(boundingBox.min.z, point.z);

	ret.max.x = maxi(boundingBox.max.x, point.x);
	ret.max.y = maxi(boundingBox.max.y, point.y);
	ret.max.z = maxi(boundingBox.max.z, point.z);

	return ret;
}

# define RND2(state) (float)hiprand_uniform(&state)
# define RND(state) (2.0*RND2(state))-1.0

#define PI 3.1415926536

int width, height;
using namespace std;
typedef unordered_map<string, double> pl;

// create an orthonormal system, assuming v1 is already normalized
DEVICE void ons(const Vec& v1, Vec& v2, Vec& v3) {
	if (std::abs(v1.x) > std::abs(v1.y)) {
		// project to the y = 0 plane and construct a normalized orthogonal vector in this plane
		float invLen = 1.f / sqrtf(v1.x * v1.x + v1.z * v1.z);
		v2 = Vec(-v1.z * invLen, 0.0f, v1.x * invLen);
	}
	else {
		// project to the x = 0 plane and construct a normalized orthogonal vector in this plane
		float invLen = 1.0f / sqrtf(v1.y * v1.y + v1.z * v1.z);
		v2 = Vec(0.0f, v1.z * invLen, -v1.y * invLen);
	}
	v3 = v1 % v2;
}

void render(Vec* pix, int s) {
	fstream myfile;
	std::cout << "iter " << s << std::endl;
	myfile.open("render.ppm", fstream::out);
	myfile << "P3\n" << width << " " << height << "\n255\n";
	for (int col = 0; col < width; col++) {
		for (int row = 0; row < height; row++) {
			int pixel_index = row * width + col;
			myfile << std::min((int)(pix[pixel_index].x / (float)(s + 1)), 255) << " ";
			myfile << std::min((int)(pix[pixel_index].y / (float)(s + 1)), 255) << " ";
			myfile << std::min((int)(pix[pixel_index].z / (float)(s + 1)), 255) << std::endl;
		}
	}
}
// Objects have color, emission, type (diffuse, specular, refractive)
// All object should be intersectable and should be able to compute their surface normals.
class Obj {
public:
	Vec cl;
	Vec emission;
	int type;
	DEVICE void setMat(Vec color_ = Vec(), Vec emission_ = 0, int type_ = 0) { cl = color_; emission = emission_; type = type_; }
	DEVICE virtual double intersect(const Ray&) const = 0;
	DEVICE virtual Vec normal(const Vec&) const = 0;
	DEVICE virtual AABB getAABB() const = 0;
};

class Plane : public Obj {
public:
	Vec n;
	double d;
	DEVICE Plane(double d_ = 0, Vec n_ = 0) {
		d = d_;
		n = n_;
	}
	DEVICE double intersect(const Ray& ray) const {
		double d0 = n.dot(ray.d);
		if (d0 != 0) {
			double t = -1 * (((n.dot(ray.o)) + d) / d0);
			return (t > eps) ? t : 0;
		}
		else return 0;
	}
	DEVICE Vec normal(const Vec& p0) const { return n; }
	DEVICE AABB getAABB() const {
		if (n.x == 0 && n.y == 0) return AABB(Vec(-inf, -inf, d * n.z), Vec(inf, inf, d * n.z));
		if (n.x == 0 && n.z == 0) return AABB(Vec(-inf, d * n.y, -inf), Vec(inf, d * n.y, inf));
		if (n.y == 0 && n.z == 0) return AABB(Vec(d * n.x, -inf, -inf), Vec(d * n.x, inf, inf));
		return AABB(Vec(-inf, -inf, -inf), Vec(inf, inf, inf));
	}
};

class Sphere : public Obj {
public:
	Vec c;
	double r;

	DEVICE Sphere(double r_ = 0, Vec c_ = 0) { c = c_; r = r_; }
	DEVICE double intersect(const Ray& ray) const {
		double b = ((ray.o - c) * 2).dot(ray.d);
		double c_ = (ray.o - c).dot((ray.o - c)) - (r * r);
		double disc = b * b - 4 * c_;
		if (disc < 0) return 0;
		else disc = sqrt(disc);
		double sol1 = -b + disc;
		double sol2 = -b - disc;
		return (sol2 > eps) ? sol2 / 2 : ((sol1 > eps) ? sol1 / 2 : 0);
	}

	DEVICE Vec normal(const Vec& p0) const {
		return (p0 - c).norm();
	}
	DEVICE AABB getAABB() const {
		return AABB(Vec(c.x - r, c.y - r, c.z - r), Vec(c.x + r, c.y + r, c.z + r));
	}
};

class Intersection {
public:
	DEVICE Intersection() { t = inf; object = nullptr; }
	DEVICE Intersection(double t_, Obj* object_) { t = t_; object = object_; }
	DEVICE operator bool() { return object != nullptr; }
	double t;
	Obj* object;
};

class Scene {
	Obj** objects;
	int list_size;

public:

	DEVICE Scene() {}

	DEVICE Scene(Obj** objs, int size) {
		objects = objs;
		list_size = size;
	}

	DEVICE Intersection intersect(const Ray& ray) const {
		Intersection closestIntersection;
		// intersect all objects, one after the other
		for (int i = 0; i < list_size; i++) {
			Intersection inter = Intersection(objects[i]->intersect(ray), objects[i]);
			if (inter.t > eps && inter.t < closestIntersection.t) {
				closestIntersection = inter;
			}
		}

		return closestIntersection;
	}
};

class 	Data {
public:
	int type;
	Vec clr;
	double cost;
	Vec emission;

	DEVICE Data(int t = 0, Vec cl = Vec(0), double co = 0, Vec em = Vec(0)) { type = t; clr = cl; cost = co, emission = em; }
};

// Class for generating the Halton low-discrepancy series for Quasi
// Monte Carlo integration.
class Halton {
	double value, inv_base;
public:
	DEVICE void number(int i, int base) {
		double f = inv_base = 1.0 / base;
		value = 0.0;
		while (i > 0) {
			value += f * (double)(i % base);
			i /= base;
			f *= inv_base;
		}
	}
	DEVICE void next() {
		double r = 1.0 - value - 0.0000001;
		if (inv_base < r) value += inv_base;
		else {
			double h = inv_base, hh;
			do { hh = h; h *= inv_base; } while (h >= r);
			value += hh + h - 1.0;
		}
	}
	DEVICE double get() { return value; }
};

// Input is the pixel offset, output is the appropriate coordinate
// on the image plane
DEVICE Vec camcr(const double x, const double y, int width, int height) {
	double w = width;
	double h = height;
	float fovx = PI / 4;
	float fovy = (h / w) * fovx;
	return Vec(((2 * x - w) / w) * tan(fovx),
		-((2 * y - h) / h) * tan(fovy),
		-1.0);
}

// Uniform sampling on a hemisphere to produce outgoing ray directions.
// courtesy of http://www.rorydriscoll.com/2009/01/07/better-sampling/
DEVICE Vec hemisphere(double u1, double u2) {
	const double r = sqrt(1.0 - u1 * u1);
	const double phi = 2 * PI * u2;
	return Vec(cos(phi) * r, sin(phi) * r, u1);
}

__device__ void trace(Data* clrlist, Ray& ray, Scene** scene, Vec& clr, float& refr_ind, const int bounce_max, hiprandState localState, int penetration_index) {
	// Russian roulette: starting at depth 5, each recursive step will stop with a probability of 0.1
	Data dt;
	Vec tmp;
	Intersection intersection;
	Vec hp;
	Vec N;
	Vec rotX, rotY;
	Vec sampledDir;
	Vec rotatedDir;

	int iter = 0;
	double cost;
	double rrFactor = 1.0;
	double n;
	double R0;
	double cost1;
	double cost2;
	double Rprob;
	const double rrStopProbability = 0.1;
	for (int depth = 0; depth < bounce_max; depth++) {
		if (depth >= 5) {
			if ((RND2(localState)) <= rrStopProbability) {
				break;
			}
			rrFactor = 1.0 / (1.0 - rrStopProbability);
		}

		intersection = (*scene)->intersect(ray);

		if (!intersection) break;

		// Travel the ray to the hit point where the closest object lies and compute the surface normal there.
		hp = ray.o + ray.d * intersection.t;
		N = intersection.object->normal(hp);
		ray.o = hp;

		// Add the emission, the L_e(x,w) part of the rendering equation, but scale it with the Russian Roulette
		// probability weight.
		const Vec emission = intersection.object->emission;
		tmp = emission * rrFactor;

		// Diffuse BRDF - choose an outgoing direction with hemisphere sampling.
		if (intersection.object->type == 1) {
			ons(N, rotX, rotY);
			sampledDir = hemisphere((RND2(localState)), (RND2(localState)));
			rotatedDir.x = Vec(rotX.x, rotY.x, N.x).dot(sampledDir);
			rotatedDir.y = Vec(rotX.y, rotY.y, N.y).dot(sampledDir);
			rotatedDir.z = Vec(rotX.z, rotY.z, N.z).dot(sampledDir);
			ray.d = rotatedDir;	// already normalized
			cost = ray.d.dot(N);
			dt = Data(1, intersection.object->cl, cost, tmp);
		}

		// Specular BRDF - this is a singularity in the rendering equation that follows
		// delta distribution, therefore we handle this case explicitly - one incoming
		// direction -> one outgoing direction, that is, the perfect reflection direction.
		if (intersection.object->type == 2) {
			cost = ray.d.dot(N);
			ray.d = (ray.d - N * (cost * 2)).norm();
			dt = Data(2, intersection.object->cl, cost, tmp);
		}

		// Glass/refractive BRDF - we use the vector version of Snell's law and Fresnel's law
		// to compute the outgoing reflection and refraction directions and probability weights.
		if (intersection.object->type == 3) {
			n = refr_ind;
			R0 = (1.0 - n) / (1.0 + n);
			R0 = R0 * R0;
			if (N.dot(ray.d) > 0) { // we're inside the medium
				N = N * -1;
				n = 1 / n;
			}
			n = 1 / n;
			cost1 = (N.dot(ray.d)) * -1; // cosine of theta_1
			cost2 = 1.0 - n * n * (1.0 - cost1 * cost1); // cosine of theta_2
			Rprob = R0 + (1.0 - R0) * pow(1.0 - cost1, 5.0); // Schlick-approximation
			if (cost2 > 0 && (RND2(localState)) > Rprob) { // refraction direction
				ray.d = ((ray.d * n) + (N * (n * cost1 - sqrt(cost2)))).norm();
			}
			else { // reflection direction
				ray.d = (ray.d + N * (cost1 * 2)).norm();
			}
			dt = Data(3, intersection.object->cl, cost1, tmp);
		}

		clrlist[bounce_max - depth - 1] = dt;
		iter++;
	}
	for (int i = bounce_max - iter; i < bounce_max; i++) {
		if (clrlist[i].type == 1) {
			clr = clrlist[i].emission + (clr * clrlist[i].clr) * clrlist[i].cost * 0.1 * rrFactor;
		}
		if (clrlist[i].type == 2) {
			clr = clrlist[i].emission + clr * rrFactor;
		}
		if (clrlist[i].type == 3) {
			if (i == bounce_max - 1 || (i - 2 >= 0 && clrlist[i - 2].emission != Vec(0)))
				clr = clrlist[i].emission + (clr * clrlist[i].clr) * 1.15 * rrFactor;
			else
				clr = clrlist[i].emission + clr * 1.15 * rrFactor;
		}
	}
}


__global__ void calc_render(int spt, int bounce_max, Data* clrlist, float refr_ind, int spp, Scene** scene, Vec* pix, hiprandState* rand_state, int actual, int width, int height, int penetration_index) {
	int row = threadIdx.x + blockIdx.x * blockDim.x;
	int col = threadIdx.y + blockIdx.y * blockDim.y;
	if ((col >= width) || (row >= height)) return;
	int pixel_index = col * width + row;
	hiprand_init((double)((actual + 1) * pixel_index), pixel_index, 0, &rand_state[pixel_index]);
	for (int s = 0; s < spt; s++) {
		//hiprandState localState = rand_state[pixel_index];
		Vec clr = Vec(0, 0, 0);
		Ray ray;
		ray.o = (Vec(0, 0, 0)); // rays start out from here
		Vec cam = camcr(col, row, width, height); // construct image plane coordinates
		cam.x = cam.x + (RND(rand_state[pixel_index])) / 700; // anti-aliasing for free
		cam.y = cam.y + (RND(rand_state[pixel_index])) / 700;
		ray.d = (cam - ray.o).norm(); // point from the origin to the camera plane
		trace(&(clrlist[pixel_index * bounce_max]), ray, scene, clr, refr_ind, bounce_max, rand_state[pixel_index], penetration_index);
		pix[pixel_index] = pix[pixel_index] + clr;
	}
}

__global__ void create_world(Obj** d_list, int size, Scene** d_scene) {
	d_list[0] = new Sphere(1.05, Vec(-0.75, -1.45, -4.4));
	d_list[0]->setMat(Vec(4, 8, 4), Vec(0), 2);

	d_list[1] = new Sphere(0.5, Vec(2.0, -2.05, -3.7));
	d_list[1]->setMat(Vec(1, 1, 1), Vec(0), 3);

	d_list[2] = new Sphere(0.6, Vec(-1.75, -1.95, -3.1));
	d_list[2]->setMat(Vec(4, 4, 12), Vec(0), 1);

	d_list[3] = new Plane(2.5, Vec(0, 1, 0));
	d_list[3]->setMat(Vec(6, 6, 6), Vec(0), 1);

	d_list[4] = new Plane(5.5, Vec(0, 0, 1));
	d_list[4]->setMat(Vec(6, 6, 6), Vec(0), 1);

	d_list[5] = new Plane(2.75, Vec(1, 0, 0));
	d_list[5]->setMat(Vec(10, 2, 2), Vec(0), 1);

	d_list[6] = new Plane(2.75, Vec(-1, 0, 0));
	d_list[6]->setMat(Vec(2, 10, 2), Vec(0), 1);

	d_list[7] = new Plane(3.0, Vec(0, -1, 0));
	d_list[7]->setMat(Vec(6, 6, 6), Vec(0), 1);

	d_list[8] = new Plane(0.5, Vec(0, 0, -1));
	d_list[8]->setMat(Vec(6, 6, 6), Vec(0), 1);

	d_list[9] = new Sphere(0.5, Vec(0, 0.5, -2));
	d_list[9]->setMat(Vec(2, 2, 10), Vec(0), 2);

	d_list[10] = new Sphere(0.5, Vec(0, 1.9, -3));
	d_list[10]->setMat(Vec(2, 2, 10), Vec(5000, 2000, 2000), 1);

	*d_scene = new Scene(d_list, size);
}

void render(int id, int size, int spp, double refr_index, int spt) {
	srand(time(NULL));
	int tx = 16;
	int ty = 16;
	int bounce_max = 7;
	int obj_num = 11;
	int penetration_index = 1;

	Obj** list;
	hipMalloc((void**)&list, obj_num * sizeof(Obj*));
	Scene** scene;
	hipMalloc((void**)&scene, sizeof(Scene*));

	create_world<<<1, 1>>>(list, obj_num, scene);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	width = size;
	height = size;

	dim3 blocks(width / tx + 1, height / ty + 1);
	dim3 threads(tx, ty);
	
	Vec* d_pix;
	hipMalloc((void**)&d_pix, width * height * sizeof(Vec));
	Vec* h_pix = (Vec*)malloc(width * height * sizeof(Vec));
	
	hiprandState* d_rand_state;
	hipMalloc((void**)&d_rand_state, width * height * sizeof(hiprandState));
	Data* clrlist;
	hipMalloc((void**)&clrlist, sizeof(Data) * bounce_max * width * height);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	// correlated Halton-sequence dimensions
	Halton hal, hal2;
	hal.number(0, 2);
	hal2.number(0, 2);

	bool running = true;

	for (int s = 0; s < spp; s += spt) {
		if (running) {
			calc_render<<<blocks, threads>>>(spt, bounce_max, clrlist, refr_index, spp, scene, d_pix, d_rand_state, s, width, height, penetration_index);
			checkCudaErrors(hipGetLastError());
			checkCudaErrors(hipDeviceSynchronize());
		}
		if (!running) return;
		hipMemcpy(h_pix, d_pix, width * height * sizeof(Vec), hipMemcpyDeviceToHost);
		render(h_pix, s + spt);
	}
	hipFree(d_rand_state);
}

int main() {
	render(1, 600, 1000, 1.5, 100);
}
