#include "hip/hip_runtime.h"

#include "tinyRt.hpp"
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include<time.h>

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__
void trace(Data *clrlist, Ray &ray, Scene **scene, Vector3& clr, float &refr_ind, const int bounce_max, hiprandState localState) {
	// Russian roulette: starting at depth 5, each recursive step will stop with a probability of 0.1
	Data dt;
	Vector3 tmp;
	Intersection intersection;
	Vector3 hp;
	Vector3 N;
	Vector3 rotX, rotY;
	Vector3 sampledDir;
	Vector3 rotatedDir;

	int iter = 0;
	double cost;
	double rrFactor = 1.0;
	double n;
	double R0;
	double cost1;
	double cost2;
	double Rprob;
	const double rrStopProbability = 0.1;
	for (int depth = 0; depth < bounce_max; depth++) {
		if (depth >= 5) {
			if ((RND2(localState)) <= rrStopProbability) {
				break;
			}
			rrFactor = 1.0 / (1.0 - rrStopProbability);
		}

		intersection = (*scene)->intersect(ray);
		
		if (!intersection) break;
		
		// Travel the ray to the hit point where the closest object lies and compute the surface normal there.
		hp = ray.o + ray.d * intersection.t;
		N = intersection.obj->normal(hp);
		ray.o = hp;

		// Add the emission, the L_e(x,w) part of the rendering equation, but scale it with the Russian Roulette
		// probability weight.
		const Vector3 emission = intersection.obj->emission;
		tmp = emission * rrFactor;

		// Diffuse BRDF - choose an outgoing direction with hemisphere sampling.
		if (intersection.obj->type == 1) {
			ons(N, rotX, rotY);
			sampledDir = hemisphere((RND2(localState)), (RND2(localState)));
			rotatedDir.x = Vector3(rotX.x, rotY.x, N.x).dot(sampledDir);
			rotatedDir.y = Vector3(rotX.y, rotY.y, N.y).dot(sampledDir);
			rotatedDir.z = Vector3(rotX.z, rotY.z, N.z).dot(sampledDir);
			ray.d = rotatedDir;	// already normalized
			cost = ray.d.dot(N);
			dt = Data(1, intersection.obj->color, cost, tmp);
		}
		
		// Specular BRDF - this is a singularity in the rendering equation that follows
		// delta distribution, therefore we handle this case explicitly - one incoming
		// direction -> one outgoing direction, that is, the perfect reflection direction.
		if (intersection.obj->type == 2) {
			cost = ray.d.dot(N);
			ray.d = (ray.d - N*(cost * 2)).norm();
			dt = Data(2, intersection.obj->color, cost, tmp);
		}
		
		// Glass/refractive BRDF - we use the vector version of Snell's law and Fresnel's law
		// to compute the outgoing reflection and refraction directions and probability weights.
		if (intersection.obj->type == 3) {
			n = refr_ind;
			R0 = (1.0 - n) / (1.0 + n);
			R0 = R0*R0;
			if (N.dot(ray.d) > 0) { // we're inside the medium
				N = N*-1;
				n = 1 / n;
			}
			n = 1 / n;
			cost1 = (N.dot(ray.d))*-1; // cosine of theta_1
			cost2 = 1.0 - n*n*(1.0 - cost1*cost1); // cosine of theta_2
			Rprob = R0 + (1.0 - R0) * pow(1.0 - cost1, 5.0); // Schlick-approximation
			if (cost2 > 0 && (RND2(localState)) > Rprob) { // refraction direction
				ray.d = ((ray.d*n) + (N*(n*cost1 - sqrt(cost2)))).norm();
			} else { // reflection direction
				ray.d = (ray.d + N*(cost1 * 2)).norm();
			}
			dt = Data(3, intersection.obj->color, cost1, tmp);
		}

		clrlist[bounce_max - depth - 1] = dt;
		iter++;
	}
	for (int i = bounce_max - iter; i < bounce_max; i++) {
		if (clrlist[i].type == 1) {
			clr = clrlist[i].emission + (clr * clrlist[i].clr) * clrlist[i].cost * 0.1 * rrFactor;
		}
		if (clrlist[i].type == 2) {
			clr = clrlist[i].emission + clr * rrFactor;
		}
		if (clrlist[i].type == 3) {
			if (i == bounce_max - 1 || (i - 2 >= 0 && clrlist[i - 2].emission != Vector3(0)))
				clr = clrlist[i].emission + (clr * clrlist[i].clr) * 1.15 * rrFactor;
			else
				clr = clrlist[i].emission + clr * 1.15 * rrFactor;
		}
	}
}

__global__
void calc_render(int spt, int bounce_max, Data *clrlist, float refr_ind, int spp, Scene **scene, Vector3 *pix, hiprandState *rand_state, int actual) {
	int row = threadIdx.x + blockIdx.x * blockDim.x;
	int col = threadIdx.y + blockIdx.y * blockDim.y;
	if((col >= W) || (row >= H)) return;
	int pixel_index = col*W + row;
	hiprand_init((double)((actual + 1) * pixel_index), pixel_index, 0, &rand_state[pixel_index]);
	for (int s = 0; s < spt; s++) {
		//hiprandState localState = rand_state[pixel_index];
		Vector3 clr = Vector3(0, 0, 0);
		Ray ray;
		ray.o = (Vector3(0, 0, 0)); // rays start out from here
		Vector3 cam = camCastRay(col, row); // construct image plane coordinates
		cam.x = cam.x + (RND(rand_state[pixel_index])) / 700; // anti-aliasing for free
		cam.y = cam.y + (RND(rand_state[pixel_index])) / 700;
		ray.d = (cam - ray.o).norm(); // point from the origin to the camera plane
		trace(&(clrlist[pixel_index * bounce_max]), ray, scene, clr, refr_ind, bounce_max, rand_state[pixel_index]);
		pix[pixel_index] = pix[pixel_index] + clr;
	}
}

__global__
void create_world(Object **d_list, int size, Scene **d_scene) {
	d_list[0] = new Sphere(1.05, Vector3(-0.75, -1.45, -4.4));
	d_list[0]->setMat(Vector3(4, 8, 4), Vector3(0), 2);

	d_list[1] = new Sphere(0.5, Vector3(2.0, -2.05, -3.7));
	d_list[1]->setMat(Vector3(4, 1, 1), Vector3(0), 3);
	
	d_list[2] = new Sphere(0.6, Vector3(-1.75, -1.95, -3.1));
	d_list[2]->setMat(Vector3(4, 4, 12), Vector3(0), 1);

	d_list[3] = new Plane(2.5, Vector3(0, 1, 0));
	d_list[3]->setMat(Vector3(6, 6, 6), Vector3(0), 1);

	d_list[4] = new Plane(5.5, Vector3(0, 0, 1));
	d_list[4]->setMat(Vector3(6, 6, 6), Vector3(0), 1);

	d_list[5] = new Plane(2.75, Vector3(1, 0, 0));
	d_list[5]->setMat(Vector3(10, 2, 2), Vector3(0), 1);

	d_list[6] = new Plane(2.75, Vector3(-1, 0, 0));
	d_list[6]->setMat(Vector3(2, 10, 2), Vector3(0), 1);

	d_list[7] = new Plane(3.0, Vector3(0, -1, 0));
	d_list[7]->setMat(Vector3(6, 6, 6), Vector3(0), 1);

	d_list[8] = new Plane(0.5, Vector3(0, 0, -1));
	d_list[8]->setMat(Vector3(6, 6, 6), Vector3(0), 1);

	d_list[9] = new Sphere(0.5, Vector3(0, 1.9, -3));
	d_list[9]->setMat(Vector3(2, 2, 10), Vector3(5000, 5000, 5000), 1);
	
	*d_scene = new Scene(d_list, size);
}

int main(int ac, char **av) {
	//in av : av[1] = spp, av[2] = refraction_index
	int bounce_max = 7;
	int spt = 10000;
	int tx = 16;
	int ty = 32;
	int spp;
	float refr_ind;
	int obj_num = 10;
	float time;
	hipEvent_t start, stop;

	if (ac >= 2)
		spp = std::atoi(av[1]);
	else
		spp = 300000;

	if (ac >= 3)
		refr_ind = std::atof(av[2]);
	else
		refr_ind = 1.9;
	
	Vector3 *d_pix;
	checkCudaErrors(hipMalloc((void **)&d_pix, H*W*sizeof(Vector3)));
	Vector3 *h_pix = (Vector3 *)malloc(H*W*sizeof(Vector3));

	Object** list;
	hipMalloc((void **)&list, obj_num*sizeof(Object *));
	Scene** scene;
	hipMalloc((void **)&scene, obj_num*sizeof(Scene *));
	create_world<<<1, 1>>>(list, obj_num, scene);

	dim3 blocks(W/tx+1,H/ty+1);
	dim3 threads(tx,ty);
	Data *clrlist;
	hipMalloc((void **)&clrlist, sizeof(Data) * bounce_max * W * H);
	hiprandState *d_rand_state;
	hipMalloc((void **)&d_rand_state, W*H*sizeof(hiprandState));

	initSDL();
	for (int s = 0; s < spp; s += spt) {
		checkCudaErrors( hipEventCreate(&start) );
		checkCudaErrors( hipEventCreate(&stop) );
		checkCudaErrors( hipEventRecord(start, 0) );
		calc_render<<<blocks, threads>>>(spt, bounce_max, clrlist, refr_ind, spp, scene, d_pix, d_rand_state, s);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
		hipMemcpy(h_pix, d_pix, W*H*sizeof(Vector3), hipMemcpyDeviceToHost);
		render(h_pix, s + spt);
		checkCudaErrors( hipEventRecord(stop, 0) );
		checkCudaErrors( hipEventSynchronize(stop) );
		checkCudaErrors( hipEventElapsedTime(&time, start, stop) );
		printf("Time to generate:  %3.1f ms \n", time);
	}
	hipFree(clrlist);
	hipFree(d_rand_state);
	quitSDL();
	hipFree(d_pix);
	return 0;
}