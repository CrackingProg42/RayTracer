#include "hip/hip_runtime.h"

#include "tinyRt.hpp"

// Input is the pixel offset, output is the appropriate coordinate
// on the image plane
__device__
Vector3 camCastRay(const double x, const double y) {
	double w = W;
	double h = H;
	float fovx = PI / 4;
	float fovy = (h / w) * fovx;
	return Vector3(((2 * x - w) / w) * tan(fovx), -((2 * y - h) / h) * tan(fovy), -1.0);
}

// Uniform sampling on a hemisphere to produce outgoing ray directions
__device__
void hemisphere(double u1, double u2, Vector3 &vec) {
	const double r = sqrt(1.0 - u1*u1);
	const double phi = 2 * PI * u2;
	vec = Vector3(cos(phi)*r, sin(phi)*r, u1);
}

// given v1, set v2 and v3 so they form an orthonormal system
// (we assume v1 is already normalized)
__device__
void ons(const Vector3& v1, Vector3& v2, Vector3& v3) {
	if (std::abs(v1.x) > std::abs(v1.y)) {
		// project to the y = 0 plane and construct a normalized orthogonal vector in this plane
		float invLen = 1.f / sqrtf(v1.x * v1.x + v1.z * v1.z);
		v2 = Vector3(-v1.z * invLen, 0.0f, v1.x * invLen);
	} else {
		// project to the x = 0 plane and construct a normalized orthogonal vector in this plane
		float invLen = 1.0f / sqrtf(v1.y * v1.y + v1.z * v1.z);
		v2 = Vector3(0.0f, v1.z * invLen, -v1.y * invLen);
	}
	v3 = v1 % v2;
}