
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ int fact(int f)
{
  if (f == 0)
    return 1;
  else
    return f * fact(f - 1);
}

__global__
void execFunction(int f, int *val) {
	val[0] = fact(10);
}

int main(void)
{
	int f = 2;
	int *d_val;
	int *val = (int *)malloc(sizeof(int) * 10);
	checkCudaErrors(hipMalloc((void**)&d_val, sizeof(int) * 10));
	execFunction<<<1, 1>>>(f, d_val);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	hipMemcpy(val, d_val, sizeof(int) * 10, hipMemcpyDeviceToHost);
	std::cout << *val;
}